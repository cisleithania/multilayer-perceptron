#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include "GPU_1.h"
#include "fann.h"
#include "floatfann.h"
#include <fstream>
#include <string>
#include <sstream>
#include <iostream>
#include "f2.h"
using namespace std;


__global__ void kernel(fann *ann, fann_type *input, fann_type *b)
{
	int idx = threadIdx.x;
	b = __device__ fann_run(ann, input);
}

void useGPU::adding(fann *ann, fann_type *input, fann_type *b) {
	/*
	int *d_o_1 = 0;
	int *d_two_1 = 0;
	int *d_one_1 = 0;
	int *d_equally_1 = 0;
	int *d_percent_1 = 0;

	hipMalloc(&d_o_1, sizeof(int));
	hipMalloc(&d_two_1, sizeof(int));
	hipMalloc(&d_one_1, sizeof(int));
	hipMalloc(&d_equally_1, sizeof(int));
	hipMalloc(&d_percent_1, sizeof(int));
	
	hipMemcpy(d_o_1, &o_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_two_1, &two_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_one_1, &one_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_equally_1, &equally_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_percent_1, &percent_1, sizeof(int), hipMemcpyHostToDevice);

	testing_1 << <1, 1 >> > (d_o_1, d_one_1, d_two_1, d_equally_1, d_percent_1);

	hipMemcpy(&o_1, d_o_1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&two_1, d_two_1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&one_1, d_one_1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&equally_1, d_equally_1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&percent_1, d_percent_1, sizeof(int), hipMemcpyDeviceToHost);
	*/

	fann *d_ann = 0;
	fann_type *d_input = 0;
	fann_type *d_b = 0;

	hipMalloc((void**)&d_ann, sizeof(fann));
	hipMalloc((void**)&d_input, sizeof(fann_type));
	hipMalloc((void**)&d_b, sizeof(fann_type));

	hipMemcpy(d_ann, ann, sizeof(fann), hipMemcpyHostToDevice);
	hipMemcpy(d_input, input, sizeof(fann_type), hipMemcpyHostToDevice);
	hipMemcpy(d_input, b, sizeof(fann_type), hipMemcpyHostToDevice);

	kernel << <1, 1 >> > (d_ann, d_input, d_b);

	hipMemcpy(b, d_input, sizeof(fann_type), hipMemcpyDeviceToHost);
	
	hipFree(d_ann);
	hipFree(d_input);
	hipFree(d_b);
}
